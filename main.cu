#include"common/ArgParser.h"
#include"common/Utils.h"

#include"parallel_gpu/GNNConfig.h"

#include<iostream>

void example00(ArgParser ap){
	hipSetDevice(CUDA_DEVICE);
	gnn_actf::Sigmoid gs;
	gnn::GNeuralNetwork<float,gnn_actf::Sigmoid> s(gs);

	s.loadExamplesFromFile("data/glass.data");
	std::vector<int> layers;

	layers.push_back(9);
	layers.push_back(27);
	layers.push_back(7);

	s.setBatchSize(10);
	s.createLayers(layers);
	s.useTranspose(true);
	//s.print_weights();
	Time<millis> t;
	t.start();
	s.train();
	t.lap("Training Execution Time(ms)");

}

void example01(ArgParser ap){
	hipSetDevice(CUDA_DEVICE);
	gnn_actf::Sigmoid gs;
	gnn::GNeuralNetwork<float,gnn_actf::Sigmoid> s(gs);

	//if(!ap.exists(FIARG)) vz::error("Please provide input file!!!");
	s.loadExamplesFromFile("data/glass.data");
	std::vector<int> layers;

	layers.push_back(16);
	layers.push_back(100);
	layers.push_back(26);

	s.setBatchSize(100);
	s.createLayers(layers);
	s.useTranspose(true);
	s.setLearningRate(0.1);

	Time<millis> t;
	//s.print_weights();
	t.start();
	for(int i = 0;i<50;i++) s.train();
	t.lap("Training Execution Time(ms)");
}

void example02(ArgParser ap){
	hipSetDevice(CUDA_DEVICE);
	gnn_actf::Sigmoid gs;
	gnn::GNeuralNetwork<float,gnn_actf::Sigmoid> s(gs);

	s.loadExamplesFromFile("data/letters_train.data");
	s.loadTestExamplesFromFile("data/letters_test.data");
	std::vector<int> layers;

	layers.push_back(16);
	layers.push_back(500);
	//layers.push_back(700);
	//layers.push_back(700);
	layers.push_back(26);

	unsigned int b = ap.getUint(BARG);
	std::cout<<"b:" << b << std::endl;
	s.setBatchSize(b);
	s.createLayers(layers);
	s.useTranspose(true);
	s.setLearningRate(0.3);

	Time<millis> t;
	//s.print_weights();
	unsigned int iterations = 100;
	t.start();
	for(int i = 0;i<iterations;i++) s.train();
	s.printConfig(t.lap("Training Execution Time(ms)")/iterations);
	//s.print_weights();
	s.classify();
}

void example03(ArgParser ap){
	gnn_actf::Sigmoid gs;
	gnn::GNeuralNetwork<float,gnn_actf::Sigmoid> s(gs);

	for(int i =0 ;i<1;i++){
		//s.bench_test_kernels(MMUL,128,128,128,false);
		//s.bench_test_kernels(MMUL,1024,1024,1024,false);
		s.bench_test_kernels(BMMUL,1024,1024,1024,false);
		//s.bench_test_kernels(MMUL,2048,2048,2048,false);
		//s.bench_test_kernels(BMMUL,2048,2048,2048,false);
		s.bench_test_kernels(TMMUL,2048,2048,2048,false);
		//s.bench_test_kernels(MHPROD,2048,2048,2048, false);
		//s.bench_test_kernels(TVECPVEC,2048,2048,2048,false);
	}
}

void example04(ArgParser ap){
	gnn_actf::Sigmoid gs;
	gnn::GNeuralNetwork<float,gnn_actf::Sigmoid> s(gs);

	//gnn_actf::FSigmoid fgs;
	//gnn::GNeuralNetwork<float,gnn_actf::FSigmoid> s(fgs);

	Time<millis> t;
	t.start();
	s.loadExamplesFromFile("../mnist_train.csv");
	s.loadTestExamplesFromFile("../mnist_test.csv");
	t.lap("Read Train and Test Data");

	std::vector<int> layers;
	layers.push_back(784);//Input Layer
	layers.push_back(1024);//Hidden Layer
	layers.push_back(10);//Output Layer

	unsigned int iterations = ap.exists(IARG) ? ap.getUint(IARG) : 50 ;
	unsigned int b = ap.exists(BARG) ? ap.getUint(BARG) : 100 ;
	float r = ap.exists(DARG) ? ap.getFloat(DARG) : 0.1 ;

	s.setBatchSize(b);
	s.useTranspose(true);
	s.setLearningRate(r);
	s.createLayers(layers);
	if(!s.validateInput()) vz::error("Input + Ouput Neurons != number of features");

	std::cout<<"Training...";
	t.start();
	for(int i = 0;i<iterations;i++){ s.train(); } std::cout << std::endl;
	s.printConfig(t.lap("Training Execution Time(ms)")/iterations);

	t.start();
	std::cout<<"Computing Classification Accuracy..." << std::endl;
	s.classify();
	t.lap("Classification Elapsed Time (ms)");
}

void example05(ArgParser ap){
	gnn_actf::Sigmoid gs;
	gnn::GNeuralNetwork<float,gnn_actf::Sigmoid> s(gs);

	//gnn_actf::FSigmoid fgs;
	//gnn::GNeuralNetwork<float,gnn_actf::FSigmoid> s(fgs);

	Time<millis> t;
	t.start();
	s.loadExamplesFromFile("../mnist_train.csv");
	s.loadTestExamplesFromFile("../mnist_test.csv");
	t.lap("Read Train and Test Data");

	std::vector<int> layers;
	layers.push_back(784);//Input Layer
	layers.push_back(1024);//Hidden Layer 1
	layers.push_back(1024);//Hidden Layer 2
	layers.push_back(10);//Output Layer

	unsigned int iterations = ap.exists(IARG) ? ap.getUint(IARG) : 50 ;
	unsigned int b = ap.exists(BARG) ? ap.getUint(BARG) : 100 ;
	float r = ap.exists(DARG) ? ap.getFloat(DARG) : 0.1 ;
	std::cout<<"i:" << iterations << std::endl;
	std::cout<<"b:" << b << std::endl;
	std::cout<<"r:" << r << std::endl;

	s.setBatchSize(b);
	s.useTranspose(true);
	s.setLearningRate(r);
	s.createLayers(layers);
	if(!s.validateInput()) vz::error("Input + Ouput Neurons != number of features");

	std::cout<<"Training...";
	t.start();
	for(int i = 0;i<iterations;i++){ s.train(); } std::cout << std::endl;
	s.printConfig(t.lap("Training Execution Time(ms)")/iterations);

	t.start();
	std::cout<<"Computing Classification Accuracy..." << std::endl;
	s.classify();
	t.lap("Classification Elapsed Time");
}

int main(int argc, char **argv){
	ArgParser ap;
	ap.parseArgs(argc,argv);
	//example00(ap);
	//example01(ap);
	//example02(ap);
	int mode = ap.exists(MDARG) ? ap.getUint(MDARG) : 0 ;
	if(mode == 0) example03(ap);
	else if(mode==1) example04(ap);
	else if(mode==2) example05(ap);
}
