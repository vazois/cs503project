#include "hip/hip_runtime.h"
#include "GNNConfig.h"
#include "../common/Time.h"

#define LOAD_TILE 4
#define ACT_TILE 32
#define DELTA_TILE 4

namespace gnn_kernels{

	/*
	 * Testing activation functions on kernels.
	 */
	template<typename ACT_F>
	__global__ void bench_test_activation(ACT_F F){
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		float a = 0;
		for(int j = 0; j<blockDim.x;j++){
			a+= F.F((float)i*j);
		}
	}

	/*
	 * Initialize matrices random weights
	 */
	template<typename DATA_T>
	__global__ void randomWeights(DATA_T *W_j,unsigned int clayer, unsigned int nlayer){
		int i = blockIdx.x * blockDim.x + threadIdx.x;

		if( i < clayer * nlayer){
			//W_j[i] = cudaUniRand(i);
			//W_j[i] = i;
			W_j[i] = 0.01 * i;
		}
	}
	/*
	 * Load current batch of train examples.
	 * 		1: First layer batch array.
	 * 		2: Training example matrix.
	 * 		3: Input layer dimension
	 * 		4: Batch size dimension
	 * 		5: Offset indicating the batch being loaded.
	 * 	Notes:
	 * 		Transpose version assumes that the training examples matrix is stored
	 * 		in a row-wise manner.
	 */
	template<typename DATA_T,unsigned int TILE>
	__global__ void loadBatchT(DATA_T *A_j, DATA_T *tEx,
			unsigned int clayer, unsigned int bsize, unsigned int offset){
		__shared__ DATA_T sAj[TILE*TILE];
		int by = blockIdx.y, bx = blockIdx.x;
		int ty = threadIdx.y, tx = threadIdx.x;
		int boffset = (by * clayer + bx ) * TILE;

		sAj[ty*TILE + tx] = tEx[boffset + offset + ty * clayer + tx];
		__syncthreads();
		boffset = (bx * bsize + by ) * TILE;
		A_j[boffset + ty * bsize + tx] = sAj[tx * TILE + ty];
		__syncthreads();
	}

	template<typename DATA_T>
	__global__ void loadBatch(DATA_T *A_j, DATA_T *tEx,
			unsigned int clayer, unsigned int bsize, unsigned int offset){
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		int step = gridDim.x * blockDim.x;

		while (i < clayer * bsize){
			A_j[i] = tEx[offset + i];
			i+=step;
		}
	}

	/*
	 * Compute matrix of activation values for a single layer of a given batch.
	 *		1:	Current layer weight matrix.
	 *		2: 	Current layer matrix of activation vectors.
	 *		3: 	Next layer matrix of activation vectors.
	 *		4:	W_j = nlayer x clayer , A_j = clayer x bsize, A_jj = nlayer x bsize.
	 *		5: 	Offset: 0 for hidden and output layer, corresponding row of training example matrix for input layer.
	 */
	template<typename DATA_T, typename ACT_F, unsigned int TILE>
	__global__ void	mmul(
			DATA_T *A_jj,
			DATA_T *W_j,
			DATA_T *A_j,
			ACT_F F,
			unsigned int nlayer,
			unsigned int clayer,
			unsigned int bsize
			)
	{
		__shared__ DATA_T sWj[TILE * TILE];
		__shared__ DATA_T sAj[TILE * TILE];

		int row = ( blockIdx.y * blockDim.y + threadIdx.y );
		int col = ( blockIdx.x * blockDim.x + threadIdx.x );
		DATA_T Ajj = 0;

		int loadOffset = threadIdx.y*TILE + threadIdx.x;
		for(int i = 0;i < ((clayer - 1) / TILE) + 1; i++){
			if( row < nlayer && (i * TILE + threadIdx.x ) < clayer)
				sWj[loadOffset] = W_j[ row * clayer + i * TILE  + threadIdx.x];
			else sWj[loadOffset] = 0.0;

			if ( i*TILE + threadIdx.y < clayer && col < bsize )
				sAj[loadOffset] = A_j[(i * TILE + threadIdx.y) * bsize + col];
			else sAj[loadOffset] = 0.0;
			__syncthreads();

			for(int j = 0;j < TILE; j++){
				Ajj += sWj[threadIdx.y * TILE + j] * sAj[j * TILE + threadIdx.x];
			}
			__syncthreads();
		}
		// ( blockIdx.y * blockDim.y + threadIdx.y ) * bsize + blockIdx.x * blockDim.x + threadIdx.x
		// row * bsize + col
		if( row < nlayer && col < bsize )
			A_jj[row * bsize + col ] = Ajj;
			//A_jj[row * bsize + col ] = F.F(Ajj);
	}

	/*
	 * Kernel that computes the last layer difference between the batch activation matrix and the expected output
	 * matrix.
	 */
	template<typename DATA_T>
	__global__ void outputD(
			DATA_T *D_j,
			DATA_T *ExA_j,
			DATA_T *A_j,
			unsigned int size
		)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;

		if ( i < size){
			D_j[i] = ExA_j[i] - A_j[i];
		}

	}

	template<typename DATA_T, unsigned int TILE>
	__global__ void tmmul(
			DATA_T *D_j,
			DATA_T *W_j,
			DATA_T *D_jj,
			unsigned int clayer,
			unsigned int nlayer,
			unsigned int bsize
			)
	{
		//grid = (bsize / TILE + 1), clayer / TILE + 1
		//block = (TILE, TILE)
		__shared__ DATA_T sWj[TILE * TILE];
		__shared__ DATA_T sDjj[TILE * TILE];

		DATA_T Dj = 0.0;
		int colW = ( blockIdx.y * blockDim.y + threadIdx.x );// by * TILE + ty * clayer + threadIdx.x
		int colD = ( blockIdx.x * blockDim.x + threadIdx.x );

		int loadOffset = threadIdx.y*TILE + threadIdx.x;
		for(int i = 0; i < (nlayer - 1) / TILE + 1 ; i++){
			if( i * TILE +  threadIdx.y < nlayer && colW < clayer)
				sWj[loadOffset] = W_j[ (i * TILE +  threadIdx.y) * clayer + colW ];
			else
				sWj[loadOffset] = 0.0;

			if((i * TILE + threadIdx.y) < nlayer && colD < bsize)
				sDjj[loadOffset] = D_jj[ (i * TILE + threadIdx.y) * bsize + colD ];
			else
				sDjj[loadOffset] = 0.0;

			__syncthreads();

			for(int j=0;j<TILE;j++){
				int index = j * TILE + threadIdx.x;
				//Dj += sWj[index] * sDjj[index];
				Dj += sWj[index];
			}
			__syncthreads();
		}

		int row = ( blockIdx.y * blockDim.y + threadIdx.y );
		if( row < clayer && colD < bsize)
			D_j[row * bsize + colD] = Dj;

	}

	template<typename DATA_T>
	__global__ void printGPU(DATA_T *A, unsigned int row, unsigned int col){
		for(int i =0;i<row*col;i++){
			printf("%.2f ", A[i]);
			if((i+1)% col == 0) printf("\n");
		}
		printf("<-------------------------------------------->\n");
	}

}

namespace gnn{

	template<typename DATA_T, typename ACT_F>
	void GNeuralNetwork<DATA_T,ACT_F>::randomInit(){
		if(network == NULL) vz::error("Network architecture missing. Use createLayers first!");
		//std::cout<<"Initializing random weights: "<<std::endl;
		hipSetDevice(0);

		cudaInitRandStates();
		for(int i = 0;i < layers-1;i++){
			//std::cout<<network[i].clayer << "-->" << network[i].nlayer << std::endl;
			unsigned int vector_size = network[i].nlayer * network[i].clayer;
			dim3 grid = grid_1D(vector_size,256);
			dim3 block = block_1D(256);
			gnn_kernels::randomWeights<DATA_T><<<grid,block>>>(network[i].W_j,network[i].clayer,network[i].nlayer);
		}
	}

	template<typename DATA_T, typename ACT_F>
	void GNeuralNetwork<DATA_T,ACT_F>::train(){
		if(network == NULL) vz::error("Network architecture missing. Use createLayers first!");
		if(bsize == 0) vz::error("Batch size not set. Use setBatchSize first!");
		unsigned int nbatch = dimEx.second / this->bsize; //std::cout<< "Batch num: " << nbatch << std::endl;
		createLayerBatch();

		for(int i = 0;i < 1;i++){
			LayerBatch<DATA_T> flayer = batch[0];
			unsigned int bRow = i * this->bsize * flayer.clayer;

			dim3 lgrid((flayer.clayer-1)/LOAD_TILE + 1,(flayer.bsize - 1)/LOAD_TILE + 1,1);
			dim3 lblock(LOAD_TILE,LOAD_TILE,1);
			/*
			 * Load current batch of training examples.
			 */
			if(this->transpose){
				gnn_kernels::loadBatchT<DATA_T,LOAD_TILE><<<lgrid,lblock>>>(flayer.A_j,examples,flayer.clayer,flayer.bsize,bRow);
			}else{
				gnn_kernels::loadBatch<DATA_T><<<32,256>>>(flayer.A_j,examples,flayer.clayer,flayer.bsize,bRow);
			}
			handleDeviceErrors(hipDeviceSynchronize(),"Error executing load batch");
			//std::cout<<flayer.clayer << " x " << flayer.bsize << std::endl;
			//print_grid(lgrid,lblock);
			//gnn_kernels::printGPU<DATA_T><<<1,1>>>(flayer.A_j,flayer.clayer,flayer.bsize);

			/*
			 * Neural network feed forward step.
			 */
			//for(int k =0;k<layers;k++){
			//			printf("b(%d) = c(%d),bz(%d)\n",k,batch[k].clayer,batch[k].bsize);
			//		}
			//		printf("<<<<<)))))))))))>\n");
			for(int j = 0;j < this->layers - 1;j++){
				dim3 agrid((batch[j+1].bsize - 1)/ACT_TILE + 1, (batch[j+1].clayer - 1)/ACT_TILE + 1);
				dim3 ablock(ACT_TILE,ACT_TILE);
				gnn_kernels::mmul<DATA_T,ACT_F,ACT_TILE><<<agrid,ablock>>>
						(
								batch[j+1].A_j,
								network[j].W_j,
								batch[j].A_j,
								F,
								network[j].nlayer,
								network[j].clayer,
								batch[j].bsize
						);
				handleDeviceErrors(hipDeviceSynchronize(),"Error executing batch activation");

				/*printf(">>>>>>ACTIVATION<<<<<<< %d\n",j);
				printf("B(%d) = W(%d) * B(%d)\n",j+1,j,j);
				print_grid(agrid,ablock);
				gnn_kernels::printGPU<DATA_T><<<1,1>>>(batch[j+1].A_j,batch[j+1].clayer,batch[j+1].bsize);
				hipDeviceSynchronize();
				gnn_kernels::printGPU<DATA_T><<<1,1>>>(network[j].W_j,network[j].nlayer,network[j].clayer);
				hipDeviceSynchronize();
				gnn_kernels::printGPU<DATA_T><<<1,1>>>(batch[j].A_j,batch[j].clayer,batch[j].bsize);
				hipDeviceSynchronize();*/
			}


			/*
			 * Back propagation step.
			 */
			//for(int k =0;k<layers;k++){
			//			printf("b(%d) = c(%d),bz(%d)\n",k,batch[k].clayer,batch[k].bsize);
			//		}
			//		printf("<<<<<)))))))))))>\n");
			dim3 ogrid = grid_1D(batch[layers-1].clayer * batch[layers-1].bsize, 256);
			dim3 oblock = block_1D(256);
			gnn_kernels::outputD<DATA_T><<<ogrid,oblock>>>(
					batch[layers-1].D_j,
					batch[0].A_j,////TODO: Initialize Y matrix correctly
					batch[layers-1].A_j,
					batch[layers-1].clayer * batch[layers-1].bsize
				);
			handleDeviceErrors(hipDeviceSynchronize(),"Error executing outputD kernel");

			/*printf(">>>>>>Output Delta<<<<<<<\n");
			print_grid(ogrid,oblock);
			gnn_kernels::printGPU<DATA_T><<<1,1>>>(batch[layers-1].D_j,batch[layers-1].clayer,batch[layers-1].bsize);
			hipDeviceSynchronize();
			gnn_kernels::printGPU<DATA_T><<<1,1>>>(batch[0].A_j,batch[0].clayer,batch[0].bsize);
			hipDeviceSynchronize();
			gnn_kernels::printGPU<DATA_T><<<1,1>>>(batch[layers-1].A_j,batch[layers-1].clayer,batch[layers-1].bsize);
			hipDeviceSynchronize();*/

			/*
			 * Backpropagation transpose matrix multiplication.
			 * 		for i = layers-1 : > 1 : i--
			 * 		batch[i-1].D_j = network[i-1].W_j * batch[i].D_j
			 * 		grid = (batch[i-1].bsize / TILE + 1), batch[i-1].clayer / TILE + 1
			 * 		block = (TILE, TILE)
			 */

			//for(int k =0;k<layers;k++){
			//			printf("b(%d) = c(%d),bz(%d)\n",k,batch[k].clayer,batch[k].bsize);
			//		}
			///		printf("<<<<<)))))))))))>\n");
			for(int j = layers-1; j > 1 ; j--){
					dim3 dgrid((batch[j-1].bsize - 1) / DELTA_TILE + 1, (batch[j-1].clayer - 1) / DELTA_TILE + 1);
					dim3 dblock(DELTA_TILE, DELTA_TILE);
					printf(">>>>>>Hidden Layer Delta<<<<<<<\n");
					printf("(%d,%d,%d)\n",j-1,batch[j-1].clayer,batch[j-1].bsize);
					printf("(%d,%d,%d)\n",j-1,network[j-1].nlayer,network[j-1].clayer);
					print_grid(dgrid,dblock);

					gnn_kernels::tmmul<DATA_T,DELTA_TILE><<<dgrid,dblock>>>(
							batch[j-1].D_j,
							network[j-1].W_j,
							batch[j].D_j,
							network[j-1].clayer,
							network[j-1].nlayer,
							batch[j].bsize
							);
					handleDeviceErrors(hipDeviceSynchronize(),"Error executing tmmul kernel");

					gnn_kernels::printGPU<DATA_T><<<1,1>>>(batch[j-1].D_j,batch[j-1].clayer,batch[j-1].bsize);
					hipDeviceSynchronize();
					gnn_kernels::printGPU<DATA_T><<<1,1>>>(network[j-1].W_j,network[j-1].nlayer,network[j-1].clayer);
					hipDeviceSynchronize();
					gnn_kernels::printGPU<DATA_T><<<1,1>>>(batch[j].D_j,batch[j].clayer,batch[j].bsize);
					hipDeviceSynchronize();

					break;
			}
		}

	}

	/*
	 * Testing methods
	 */
	template<typename DATA_T, typename ACT_F>
	void GNeuralNetwork<DATA_T,ACT_F>::bench_act(){
		hipSetDevice(0);
		dim3 block(512,1,1);
		dim3 grid(128,1,1);

		/*
		 * Warm up device
		 */
		gnn_kernels::bench_test_activation<ACT_F><<<grid,block>>>(this->F);
		hipDeviceSynchronize();
		/* <END> */

		std::string msg("Benchmark ");
		msg.append(F.TAG);
		Time<millis> t;
		t.start();
		gnn_kernels::bench_test_activation<ACT_F><<<grid,block>>>(F);
		hipDeviceSynchronize();
		t.lap(msg);
	}

	template<typename DATA_T, typename ACT_F>
	void GNeuralNetwork<DATA_T,ACT_F>::print_weights(){
		DATA_T *cW_j;
		hipSetDevice(0);

		for(int i = 0;i < 1;i++){
			unsigned int size = network[i].nlayer * network[i].clayer;
			allocHostMem<DATA_T>(&cW_j,sizeof(DATA_T)*size, "Error Allocating Host Weight Matrix");
			safeCpyToHost<DATA_T>(cW_j,network[i].W_j,sizeof(DATA_T)*size, "Error Allocating Copying Weight Matrix From Device");

			for(int j = 0;j<size;j++){
				std::cout<<cW_j[j] << " ";
				if((j+1)%network[i].clayer == 0) std::cout<<std::endl;
			}
			std::cout<<std::endl;
		}

	}

	template class GNeuralNetwork<float,gnn::Sigmoid>;
	template class GNeuralNetwork<float,gnn::FSigmoid>;
	template class GNeuralNetwork<float,gnn::Arctan>;
}
